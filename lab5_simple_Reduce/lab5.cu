#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA_CHECK_RETURN(value) ((hipError_t)value != hipSuccess) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value), __LINE__, __FILE__) : printf("") 

__global__ void reduce0(double *input, double *output) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = input[i];

    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
            // printf("[%d] %lf\n", tid, sdata[tid]);
        }

        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}

double sumCpu(double *c, int size) {
    double sum;
    for (unsigned int i = 0; i < size; i++)
         sum+= c[i];
    return sum;
}

int main(void) {
    
    srand(time(NULL));

    struct timeval t1, t2;
    float elapsedTime;
    hipEvent_t start, stop;

    double *h_in;
    double *h_out;
    double *d_in;
    double *d_out;

    int numThreadsPerBlock = 1024;
    long numInputElements = 2<<20; // 2<<20 .. 2 << 25
    int numOutputElements = (numInputElements + numThreadsPerBlock - 1) / numThreadsPerBlock;

    fprintf(stdout, "N %d block %d numThreadsPerBlock %d\n", numInputElements, numOutputElements, numThreadsPerBlock);


    h_in = (double *)malloc(numInputElements * sizeof(double));
    h_out = (double *)malloc(numOutputElements * sizeof(double));


    for (int i = 0; i < numInputElements; i++) {
        h_in[i] = 1.0;
    }

    const dim3 blockSize(numThreadsPerBlock, 1, 1);
    const dim3 gridSize(numOutputElements, 1, 1);

    CUDA_CHECK_RETURN(hipMalloc((void **)&d_in, numInputElements * sizeof(double)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_out, numOutputElements * sizeof(double)));

    CUDA_CHECK_RETURN(hipMemcpy(d_in, h_in, numInputElements * sizeof(double), hipMemcpyHostToDevice));
    

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));

    CUDA_CHECK_RETURN(hipEventRecord(start, 0));

    reduce0 <<< gridSize, blockSize, numThreadsPerBlock*sizeof(double) >>> (d_in, d_out);
    
    CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));

    CUDA_CHECK_RETURN(hipGetLastError());
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));

    CUDA_CHECK_RETURN(hipEventDestroy(start));
    CUDA_CHECK_RETURN(hipEventDestroy(stop));    
    
    CUDA_CHECK_RETURN(hipMemcpy(h_out, d_out, numOutputElements * sizeof(double), hipMemcpyDeviceToHost));

    double sumGPU = 0.0;

    for (int i = 0; i < numOutputElements; i++) {
        sumGPU += h_out[i];
        // printf("%lf\n", h_out[i]);
    }


    printf("GPU Result: %lf; elapsed time %g ms\n", sumGPU, elapsedTime);

    double sum_CPU = 0.0;

    gettimeofday(&t1, NULL);
    sum_CPU = sumCpu(h_in, numInputElements);
    gettimeofday(&t2, NULL);
    elapsedTime = (t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0)*1000.0;

    printf("CPU Result: %lf; elapsed time %g ms\n", sum_CPU, elapsedTime);

    CUDA_CHECK_RETURN(hipFree(d_in));
    CUDA_CHECK_RETURN(hipFree(d_out));


    free(h_in);
    free(h_out);

    return 0;
}
