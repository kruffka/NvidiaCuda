#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gTest(float* a)
{
    a[threadIdx.x + blockDim.x * blockIdx.x] = (float)((threadIdx.x + blockDim.x * blockIdx.x) * 2);
}

int main() {
    int m, n, k;
    scanf("%d %d %d", &m, &n, &k);

    float* mas = new float[m];
    float* da;

    int dev;
    hipSetDevice(dev); 
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    printf(" Total amount of constant memory: %lu bytes\n", deviceProp.totalConstMem);
    printf(" Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
    printf(" Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
    printf(" Warp size: %d\n", deviceProp.warpSize);
    printf(" Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf(" Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    
   
    printf("Occupancy: %g\n", (float)(k * 8) / (float)((deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize) * deviceProp.warpSize));

    hipMalloc((void**)&da, m * sizeof(float));
    gTest <<< n, k >>> (da);
    hipDeviceSynchronize();
    hipMemcpy(mas, da, m * sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = m - 4; i < m; i++)
    {
        printf("%g\n", mas[i]);
    }
    free(mas);
    hipFree(da);
 
    return 0;
}