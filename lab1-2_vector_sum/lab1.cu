#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void sum(float* a, float* b, float* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}


#define CUDA_CHECK_RETURN(value) ((hipError_t)value != 0) ? printf("Error %s at line %d in the file %s\n", hipGetErrorString((hipError_t)value),  __LINE__, __FILE__) : printf("") 

int main() { 

    float *a, *b, *c;

    float *d_a, *d_b, *d_c;

    float elapsedTime;
    hipEvent_t start, stop;

    FILE *file = fopen("./results", "w");
    if (file == NULL) {
        printf("error opening file\n");
        exit(0); 
    }

    for (int threads = 1; threads < 1024; threads <<= 1) {
        for (int N = 1 << 10; N < (1 << 23); N <<= 1) {
            int block = (N + threads - 1) / threads;
            printf("thread %d N %d block %d\n", threads, N, block);

            a = new float[block * threads];
            b = new float[block * threads];
            c = new float[block * threads];

            for(int i = 0; i < block * threads; i++)
            {
                a[i] = i;
                b[i] = i;
            }


            CUDA_CHECK_RETURN(hipEventCreate(&start));
            CUDA_CHECK_RETURN(hipEventCreate(&stop));

            CUDA_CHECK_RETURN(hipMalloc((void **)&d_a, block * threads * sizeof(float)));
            CUDA_CHECK_RETURN(hipMalloc((void **)&d_b, block * threads * sizeof(float)));
            CUDA_CHECK_RETURN(hipMalloc((void **)&d_c, block * threads * sizeof(float)));

            CUDA_CHECK_RETURN(hipMemcpy(d_a, a, block * threads * sizeof(float), hipMemcpyHostToDevice));
            CUDA_CHECK_RETURN(hipMemcpy(d_b, b, block * threads * sizeof(float), hipMemcpyHostToDevice));
        
            CUDA_CHECK_RETURN(hipEventRecord(start, 0));
            sum <<< block, threads >>> (d_a, d_b, d_c);
            CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
            CUDA_CHECK_RETURN(hipEventSynchronize(stop));

            CUDA_CHECK_RETURN(hipGetLastError());

            CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
        
            fprintf(file, "elapsedTime %g block %d thread %d\n", elapsedTime, block, threads);

            CUDA_CHECK_RETURN(hipEventDestroy(start));
            CUDA_CHECK_RETURN(hipEventDestroy(stop));

            CUDA_CHECK_RETURN(hipMemcpy(c, d_c, block * threads * sizeof(float), hipMemcpyDeviceToHost));

            free(a);
            free(b);
            free(c);

            CUDA_CHECK_RETURN(hipFree(d_a));
            CUDA_CHECK_RETURN(hipFree(d_b));
            CUDA_CHECK_RETURN(hipFree(d_c));   
        }
    }
    
    fclose(file);

    return 0;
}
